#include "hip/hip_runtime.h"
//#include "base/DeviceDefs.hpp"
//#include "base/Math.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


#if 0

struct multip
{
    __host__ __device__
        int operator()(int x) { return x * 10; }
};

void hostSort(size_t N)
{
    thrust::host_vector<float> a(N);
    
    //thrust::generate(a.begin(), a.end(), []() { return (float)rand(); });
    thrust::generate(a.begin(), a.end(), rand);

    thrust::sort(a.begin(), a.end());

    //for each (auto var in a) {
    //    std::cout << var << ' ';
    //}
    //std::cout << '\n';
}

float testReduce(size_t N)
{
    thrust::host_vector<float> a(N);
    thrust::device_vector<float> b(N);
    thrust::device_vector<float> c(N);

    //thrust::generate(a.begin(), a.end(), []() { return (float)rand(); });
    //thrust::generate(b.begin(), b.end(), rand);

    // Copy to device
    // b = a;

    //thrust::transform(a.begin(), a.end(), b.begin(), );
    //thrust::transform(b.begin(), b.end(), c.begin(), multip());
    thrust::transform(b.begin(), b.end(), c.begin(), [] __host__ __device__(float v) { return 10.f * v; });

    thrust::sort(c.begin(), c.end());

    // Copy to host
    //a = c;

    float res = thrust::reduce(c.begin(), c.end());

    // for each (auto var in a) {
    //     std::cout << var << ' ';
    // }
    // std::cout << '\n';

    return res;
}

float testUVM(size_t N)
{
    printf("Testing UVM\n");

    thrust::host_vector<float> a(N);
    thrust::device_vector<float> b(N);
    thrust::device_vector<float> c(N);
    thrust::host_vector<float> d(N);

    thrust::generate(a.begin(), a.end(), rand);

    float* aptr;
    hipError_t er = hipMallocManaged(&aptr, N * sizeof(float));// , hipStreamDefault);

    assert(er == hipSuccess);
    printf("aptr=0x%08llx er=%d\n", (unsigned long long)aptr, er);

    for (int i = 0; i < N; i++)
        aptr[i] = 3.14159f; // (float)rand();

    thrust::transform(b.begin(), b.end(), c.begin(), [=]  __device__(float v) { return *aptr; });

    d = c;

    for(int i=0; i<16; i++)
        std::cout << d[i] << ' ';
    std::cout << '\n';

    return 0.f;
}

#endif

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;

    C[i] = A[i] + B[i];
}

void testKernel(size_t N)
{
    thrust::device_vector<float> A(N);
    thrust::device_vector<float> B(N);
    thrust::device_vector<float> C(N);

    int numBlocks = 1;
    dim3 threadsPerBlock(32);
    VecAdd<<<numBlocks, threadsPerBlock >>>(A.data().get(), (float*)B.data().get(), (float*)C.data().get());
}

float testThrust(size_t N)
{

    testKernel(N);
    // testUVM();
    //hostSort(N);

    return 123.f;
}
