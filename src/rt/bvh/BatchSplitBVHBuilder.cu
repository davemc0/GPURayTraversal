#include "hip/hip_runtime.h"
#define FW_ENABLE_ASSERT

#include "bvh/BVHNode.hpp"
#include "bvh/BatchSplitBVHBuilder.hpp"
#include "base/Array.hpp"
#include "base/Timer.hpp"

#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>

#include <thrust/partition.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

#define BHD __device__

// Copied from Platform.hpp since I don't want to pull that into the device.
// batch processing (how many ops at the price of one)
#define n_triBatchSize    1
#define n_nodeBatchSize   1
#define n_SAHTriangleCost 1.0f
#define n_SAHNodeCost     1.0f

using FW::S32;
using FW::F32;
using FW::AABB;
using thrust::get;

S32   BHD roundToTriangleBatchSize(S32 n) { return ((n + n_triBatchSize - 1) / n_triBatchSize)*n_triBatchSize; }
S32   BHD roundToNodeBatchSize(S32 n) { return ((n + n_nodeBatchSize - 1) / n_nodeBatchSize)*n_nodeBatchSize; }

float BHD getTriangleCost(S32 n) { return roundToTriangleBatchSize(n) * n_SAHTriangleCost; }
float BHD getNodeCost(S32 n) { return roundToNodeBatchSize(n) * n_SAHNodeCost; }
float BHD getCost(int numChildNodes, int numTris) { return getNodeCost(numChildNodes) + getTriangleCost(numTris); }

void FW::BatchSplitBVHBuilder::initBBArrays(S32 maxN, FW::Scene* scene, FW::BVH& bvh)
{
    m_intArray.setManaged(true);
    m_intArray.resize(maxN * 10);
    m_boundsArray.setManaged(true);
    m_boundsArray.resize(maxN * 3);
    m_keysArray.setManaged(true);
    m_keysArray.resize(maxN * 2);

    hipMemset(m_intArray.getPtr(), 0, sizeof(S32)  * m_intArray.getSize());
    hipMemset(m_boundsArray.getPtr(), 0, sizeof(AABB) * m_boundsArray.getSize());
    hipMemset(m_keysArray.getPtr(), 0, sizeof(U64)  * m_keysArray.getSize());
    
    m_refRightIdx = m_intArray.getPtr();
    m_refLeftIdx = m_refRightIdx + maxN;
    m_refGamma = m_refLeftIdx + maxN;
    m_refSegIdx = m_refGamma + maxN;
    m_segIdxBest = m_refSegIdx + maxN; // Per-reference index into segment vector
    m_segIdxNew = m_segIdxBest + maxN;
    m_segCostBest = (F32*)(m_segIdxNew + maxN);
    m_segCostNew = m_segCostBest + maxN;
    m_segStratRefIdx = (S32*)(m_segCostNew + maxN);
    m_refBounds = m_boundsArray.getPtr();
    m_refRightBounds = m_refBounds + maxN;
    m_refLeftBounds = m_refRightBounds + maxN;
    m_refKeys = m_keysArray.getPtr();
    m_segKeys = m_refKeys + maxN;

    bvh.getTriIndices().setManaged(true);
    bvh.getTriIndices().resize(maxN);
    m_refTriIdx = bvh.getTriIndices().getPtr();

    m_tris = (const Vec3i*)scene->getTriVtxIndexBuffer().getCudaPtr();
    m_verts = (const Vec3f*)scene->getVtxPosBuffer().getCudaPtr();

    hipMemsetD32((hipDeviceptr_t)m_refGamma, FW_S32_MIN, maxN); // Set gamma to FW_S32_MIN
}

void FW::BatchSplitBVHBuilder::freeArrays()
{
    // printf("freeBBArrays()\n");
    m_intArray.reset(0);
    m_boundsArray.reset(0);
    m_keysArray.reset(0);
}

typedef thrust::tuple<AABB, AABB, S32> BBITuple; // rightBounds, leftBounds, i (absolute index)
typedef thrust::tuple<float, S32, S32> FIITuple; // Cost, leftN (index within segment), i (absolute index)

struct BoundsToCost
{
    S32* refSegIdx;
    S32* segStratRefIdx;
    S32 N, nSegments;

    BoundsToCost(S32* r, S32* s, S32 n, S32 ns) : refSegIdx(r), segStratRefIdx(s), N(n), nSegments(ns) {}

    BHD FIITuple operator()(BBITuple x)
    {
        float rightA = get<0>(x).area();
        float leftA = get<1>(x).area();
        S32 i = get<2>(x);
        S32 s = refSegIdx[i]; // Segment index in output arrays
        S32 b = FW::BatchSplitBVHBuilder::stratNumMask & segStratRefIdx[s]; // Ref index of segment start
        S32 e = (s < (nSegments - 1) ? (FW::BatchSplitBVHBuilder::stratNumMask & segStratRefIdx[s + 1]) : N) - 1;

        S32 leftN = i - b;
        S32 rightN = e - i;

        // This is just SAH of the two children. Need to add nodeSAH for it to be a full SAH. Also it's not scaled by root bounds.
        F32 childSAH = leftA * getTriangleCost(leftN) + rightA * getTriangleCost(rightN);
        // Add nodeSAH - OPT: Instead, subtract nodeSAH from leafSAH to avoid computing nodeBounds here
        AABB nodeBounds = get<0>(x) + get<1>(x);
        F32 nodeSAH = nodeBounds.area() * getNodeCost(2);
        F32 sah = childSAH + nodeSAH;
        // F32 sah = childSAH;
        //printf("i=%d s=%d b=%d e=%d sah=%f leftN=%d\n", i, s, b, e, sah, leftN);

        return thrust::make_tuple(sah, leftN, i);
    }
};

void FW::BatchSplitBVHBuilder::doGeneration(S32& N, S32& nSegments, S32 level)
{
    S32*  refTriIdx = m_refTriIdx;
    S32*  discard = m_refRightIdx; // XXX
    S32*  discard2 = m_refLeftIdx;
    S32*  refGamma = m_refGamma;
    S32*  refSegIdx = m_refSegIdx;
    S32*  segIdxBest = m_segIdxBest;
    S32*  segIdxNew = m_segIdxNew;
    F32*  segCostBest = m_segCostBest;
    F32*  segCostNew = m_segCostNew;
    S32*  segStratRefIdx = m_segStratRefIdx;
    AABB* refBounds = m_refBounds;
    AABB* refRightBounds = m_refRightBounds;
    AABB* refLeftBounds = m_refLeftBounds;
    U64*  refKeys = m_refKeys;
    U64*  segKeys = m_segKeys;

    typedef thrust::tuple<S32, S32, AABB, U64> TGBKTuple;
    typedef thrust::tuple<S32*, S32*, AABB*, U64*> TGBKItTuple;
    typedef thrust::zip_iterator<TGBKItTuple> TGBKZipIt;
    TGBKZipIt refsTGBK(thrust::make_tuple(refTriIdx, refGamma, refBounds, refKeys));

    auto OneIt = thrust::make_constant_iterator((S32)1);

    // Remove degenerates.
    // OPT: For Sweep builder move this out of the loop. If so, for speed, change it to not be a stable_partition. Split builder makes new refs.
    // OPT: Could make this part of sort predicate and use custom iterator to count how many are rejected or an atomic
    auto mid = thrust::stable_partition(thrust::device, refsTGBK, refsTGBK + N, [] BHD(const TGBKTuple r) {
        Vec3f size = get<2>(r).max() - get<2>(r).min();
        return !(min(size) < 0.0f || sum(size) == max(size));
    });

    S32 newN = get<0>(mid.get_iterator_tuple()) - refTriIdx;
    if (newN != N) printf("Removing degenerates: %d => %d\n", N, newN);
    N = newN;

    // Compute refSegIdx
    thrust::transform_inclusive_scan(thrust::device,
        thrust::make_counting_iterator((S32)0), thrust::make_counting_iterator((S32)N), refSegIdx,
        [refKeys] BHD(S32 i) { return (i == 0 || refKeys[i] == refKeys[i - 1]) ? 0 : 1; },
        [] BHD(S32 a, S32 b) { return a + b; });

    // Compute segStratRefIdx
    thrust::reduce_by_key(thrust::device,
        refKeys, refKeys + N, thrust::make_counting_iterator((S32)0),
        segKeys, segStratRefIdx,
        [] BHD(U64 ka, U64 kb) { return ka == kb; }, [] BHD(S32 a, S32 b) { return stratNone | min(a, b); });

    // Try object split in each dimension
    for (int dim = 0; dim < 3; dim++) {

        // Sort in given dimension
        thrust::sort(thrust::device,
            refsTGBK, refsTGBK + N,
            [dim, segStratRefIdx] BHD(TGBKTuple a, TGBKTuple b) {
            // S32 sa = get<4>(a); // Segment index of each reference
            // S32 sb = get<4>(b);

            S32 la = 0; // stratMask & segStratRefIdx[sa]; // sort by strategy
            S32 lb = 0; // stratMask & segStratRefIdx[sb];
            F32 ca = get<2>(a).min()[dim] + get<2>(a).max()[dim]; // centroid in dim
            F32 cb = get<2>(b).min()[dim] + get<2>(b).max()[dim];

            return (la < lb) || (la == lb && ((get<3>(a) < get<3>(b)) || (get<3>(a) == get<3>(b) && ((ca < cb || (ca == cb && get<0>(a) < get<0>(b)))))));
        });

        // Sweep right to left and determine bounds
        // refLeftBounds[i] and refRightBounds[i] contain the two AABBs for splitting at i.

        // OPT: Use transform_output iterator to just store area instead of whole AABB
        thrust::inclusive_scan_by_key(thrust::device,
            thrust::make_reverse_iterator(refKeys + N), thrust::make_reverse_iterator(refKeys),
            thrust::make_reverse_iterator(refBounds + N), thrust::make_reverse_iterator(refRightBounds + N),
            [] BHD(U64 ka, U64 kb) { return ka == kb; },
            [] BHD(AABB a, AABB b) { return a + b; });

        // Sweep left to right and determine bounds
        thrust::exclusive_scan_by_key(thrust::device,
            refKeys, refKeys + N, refBounds, refLeftBounds, AABB(),
            [] BHD(U64 ka, U64 kb) { return ka == kb; },
            [] BHD(AABB a, AABB b) { return a + b; });

        // OPT: Store a segment's full AABB into its final BVHNode, since we know its location now

        // Select lowest SAH; BBI -> FII
        auto BoundsIt(thrust::make_zip_iterator(thrust::make_tuple(refRightBounds, refLeftBounds, thrust::make_counting_iterator((S32)0)))); // BBI

        // OPT: Only need to write the keys out once. Could use discard_iterator on the other two dimensions.
        // OPT: discard is unneeded; should use a discard_iterator to get rid of discard output, but was getting errors.
        auto segValues = thrust::make_zip_iterator(thrust::make_tuple(dim == 0 ? segCostBest : segCostNew, dim == 0 ? segIdxBest : segIdxNew, discard)); // FII

        auto segEnd = thrust::reduce_by_key(thrust::device,
            refKeys, refKeys + N, thrust::make_transform_iterator(BoundsIt, BoundsToCost(refSegIdx, segStratRefIdx, N, nSegments)),
            segKeys, segValues, // OPT: I don't use segKeys. Should use discard_iterator, but need nSegments.
            [] BHD(U64 ka, U64 kb) { return ka == kb; },
            [] BHD(FIITuple a, FIITuple b) { return get<0>(a) < get<0>(b) ? a : // Reduction operator is min cost
                (get<0>(a) > get<0>(b) ? b :
                (abs(get<1>(a) - get<2>(a)) < abs(get<1>(b) - get<2>(b)) ? a : b)); }); // Tie break in the middle

        nSegments = segEnd.first - segKeys;
        U64 segKeys0 = segKeys[0];
        S32 thisStrategy = stratObjectSplit | (dim << stratBitOffset);

        printf("Alevel=%d dim=%d nSegments=%d segKeys[0]=%016llx\n", level, dim, nSegments, segKeys0);
        if (level > -90) {
            hipDeviceSynchronize(); // XXX
            for (int s = 0; s < nSegments; s++) {
                printf("As=%d Strat=0x%x RefIdx=%d IdxBest=%d CostBest=%f IdxNew=%d CostNew=%f Keys=%016llx\n", s, (U32)segStratRefIdx[s] >> stratBitOffset,
                    stratNumMask & segStratRefIdx[s], segIdxBest[s], segCostBest[s], segIdxNew[s], segCostNew[s], segKeys[s]);
            }
        }

        // Update best strategy
        // OPT: Would rather do this as a conditional_iterator as part of reduce_by_key.
        if (dim > 0)
            thrust::for_each_n(thrust::device, thrust::counting_iterator<S32>((S32)0), nSegments, [=] BHD(S32 i) {
            if (segCostNew[i] < segCostBest[i]) {
                segCostBest[i] = segCostNew[i];
                segIdxBest[i] = segIdxNew[i];
                segStratRefIdx[i] = thisStrategy | (stratNumMask & segStratRefIdx[i]);
            }
        });

        printf("Blevel=%d dim=%d nSegments=%d segKeys[0]=%016llx\n", level, dim, nSegments, segKeys0);
        if (level > -90) {
            hipDeviceSynchronize(); // XXX
            for (int s = 0; s < nSegments; s++) {
                printf("Bs=%d Strat=0x%x RefIdx=%d IdxBest=%d CostBest=%f IdxNew=%d CostNew=%f Keys=%016llx\n", s, (U32)segStratRefIdx[s] >> stratBitOffset,
                    stratNumMask & segStratRefIdx[s], segIdxBest[s], segCostBest[s], segIdxNew[s], segCostNew[s], segKeys[s]);
            }
        }
    }

    // Compute each segment's leaf SAH and update best strategy
    S32 maxLeafSize = m_platform.getMaxLeafSize(), minLeafSize = m_platform.getMinLeafSize();
    thrust::for_each_n(thrust::device, thrust::counting_iterator<S32>((S32)0), nSegments, [=] BHD(S32 s) {
        S32 ind = stratNumMask & segStratRefIdx[s];
        S32 leafN = (s < (nSegments - 1) ? (stratNumMask & segStratRefIdx[s + 1]) : N) - 1; // OPT: Add a fake segment whose index is N to simplify this

        F32 leafSAH = FW_F32_MAX;
        if (leafN <= minLeafSize)
            leafSAH = FW_F32_MIN;
        else if (leafN <= maxLeafSize) {
            AABB& bounds = refRightBounds[ind];
            leafSAH = bounds.area() * getTriangleCost(leafN);
        }

        if (leafSAH < segCostBest[s]) {
            segCostBest[s] = leafSAH;
            segIdxBest[s] = FW_S32_MAX;
            segStratRefIdx[s] = stratLeaf | ind;
        }
    });

    if (level > -90) {
        hipDeviceSynchronize(); // XXX
        printf("Clevel=%d nSegments=%d\n", level, nSegments);
        for (int s = 0; s < nSegments; s++) {
            printf("Cs=%d Strat=0x%x RefIdx=%d IdxBest=%d CostBest=%f IdxNew=%d CostNew=%f Keys=%016llx\n", s, (U32)segStratRefIdx[s] >> stratBitOffset,
                stratNumMask & segStratRefIdx[s], segIdxBest[s], segCostBest[s], segIdxNew[s], segCostNew[s], segKeys[s]);
        }

        hipDeviceSynchronize(); // XXX
        for (int i = 0; i < N; i++)
            printf("Ci=%d refSegIdx[i]=%d refKeys[i]=%016llx\n", i, refSegIdx[i], refKeys[i]);
    }

    // Count how many refs want each kind of strategy to give me indices to them after they're sorted
    // thrust::inclusive_scan with an output tuple with a value per strategy. Could fold it into the for_each_n and use atomic counters?

    // Sort each segment by its best dimension
    typedef thrust::tuple<S32, S32, AABB, U64, S32>    TGBKITuple;
    typedef thrust::tuple<S32*, S32*, AABB*, U64*, S32*> TGBKIItTuple;
    typedef thrust::zip_iterator<TGBKIItTuple> TGBKIZipIt;
    TGBKIZipIt refsTGBKI(thrust::make_tuple(refTriIdx, refGamma, refBounds, refKeys, refSegIdx));
    // XXX Need to make sure that for multi-reference leaves the right ref sorts to the end to make gamma work.

    // OPT: Sort by strat to give good spans for doing separate algorithms in next pass; maybe lets us keep a sorted array per dim
    // OPT: Could store strat in 3 msbs of segKeys instead of segRefIdx
    thrust::sort(thrust::device,
        refsTGBKI, refsTGBKI + N, [segStratRefIdx] BHD(TGBKITuple a, TGBKITuple b) {
        S32 sa = get<4>(a); // Segment index of each reference
        S32 sb = get<4>(b);

        int dim = (stratDimMask & segStratRefIdx[sa]) >> stratBitOffset;
        S32 la = 0; // stratMask & segStratRefIdx[sa]; // sort by strategy
        S32 lb = 0; // stratMask & segStratRefIdx[sb];
        F32 ca = get<2>(a).min()[dim] + get<2>(a).max()[dim]; // centroid in dim
        F32 cb = get<2>(b).min()[dim] + get<2>(b).max()[dim];

        return (la < lb) || (la == lb && ((get<3>(a) < get<3>(b)) || (get<3>(a) == get<3>(b) && ((ca < cb || (ca == cb && get<0>(a) < get<0>(b)))))));
    });

    if (level > -90) {
        hipDeviceSynchronize(); // XXX
        for (int i = 0; i < N; i++)
            printf("Di=%d refSegIdx[i]=%d refKeys[i]=%016llx\n", i, refSegIdx[i], refKeys[i]);
    }
    // Update Nactive here so only the active ones get their keys updated

    // Try to get rid of keys and just use segIdx. Have to be able to put them back in order to make gamma work.
    // XXX Will splits screw up gamma by inserting nodes between index and what it points to?

    // Update keys to partition each segment at the best location
    thrust::for_each(thrust::device, thrust::make_counting_iterator((S32)0), thrust::make_counting_iterator((S32)N),
        [=] BHD(S32 i) {
        S32 s = refSegIdx[i]; // Segment index in output arrays
        S32 b = stratNumMask & segStratRefIdx[s]; // Ref index of segment start
        S32 e = (s < (nSegments - 1) ? (stratNumMask & segStratRefIdx[s + 1]) : N) - 1; // OPT: Add a fake segment whose index is N to simplify this
        //printf("i=%d s=%d b=%d e=%d segIdxBest[s]=%d\n", i, s, b, e, segIdxBest[s]);

        if (i - b >= segIdxBest[s])
            // My offset within segment is to the right of the split index
            refKeys[i] = refKeys[i] | 1ull << (U64)(63 - level);

        // If I'm at the start or end of the segment and the gamma slot hasn't been claimed yet so I record the relative split location.
        if (i == b && refGamma[i] == FW_S32_MIN)
            refGamma[i] = segIdxBest[s];
        if (i == e && refGamma[i] == FW_S32_MIN)
            refGamma[i] = segIdxBest[s] - (e - b); // The (negative) offset from i to the relative split location.
    });

    if (level > -90) {
        hipDeviceSynchronize(); // XXX
        for (int s = 0; s < nSegments; s++) {
            printf("Es=%d Strat=0x%x RefIdx=%d IdxBest=%d CostBest=%f IdxNew=%d CostNew=%f Keys=%016llx\n", s, (U32)segStratRefIdx[s] >> stratBitOffset,
                stratNumMask & segStratRefIdx[s], segIdxBest[s], segCostBest[s], segIdxNew[s], segCostNew[s], segKeys[s]);
        }

        hipDeviceSynchronize(); // XXX
        for (int i = 0; i < N; i++)
            printf("Ei=%d refSegIdx[i]=%d refKeys[i]=%016llx\n", i, refSegIdx[i], refKeys[i]);
    }

    printf("Done with generation %d.\n", level);
}

FW::BVHNode* FW::BatchSplitBVHBuilder::makeNodes(S32 N)
{
    printf("makeNodes\n");
    S32*  gamma = m_refGamma;
    AABB* refBounds = m_refBounds;

    // In parallel, make all the leaves
    hipDeviceSynchronize(); // Needed to allocate managed with ArrayAllocator.
    LeafNode* leaves = new LeafNode[N];
    InnerNode* inner = new InnerNode[N];

    // OPT: Store refBounds directly in BVHNodes. Or replace BVHNodes with SOA.

    thrust::for_each(thrust::device, thrust::make_counting_iterator((S32)0), thrust::make_counting_iterator((S32)N),
        [=] BHD(S32 i) {
        // Fill leaf node i
        leaves[i].m_bounds = refBounds[i];
        leaves[i].m_lo = i;
        leaves[i].m_hi = i + 1;
    });

    // Object splits:
    // Split location [i] means there are i refs to the left and N - i refs to the right. (=> [0] is unused.)
    // It means a split between [i-1] and [i]. (Different than Karras 2012.)

    // gamma[i] is the offset from i to the split of the segment that either starts or ends at i.
    // If that index is a segment of length 1 then the child is in Leaves; otherwise it's in Inner.
    // gamma[i] > 0 if i's segment is to the right; <= 0 if to the left.

    thrust::for_each(thrust::device, thrust::make_counting_iterator((S32)0), thrust::make_counting_iterator((S32)N - 1),
        [=] BHD(S32 i) {
        // Fill inner node i
        bool leftIsLeaf, rightIsLeaf;
        S32 dj = gamma[i];
        S32 j = i + dj;
        if (dj <= 0) { // am a left child
            leftIsLeaf = gamma[j - 1] > 0; // true if the span starting at j-1 is a child of i's span, not an ancestor
            rightIsLeaf = dj == 0;
        }
        else { // am a right child
            leftIsLeaf = dj == 1;
            rightIsLeaf = gamma[j] <= 0;
        }

        BVHNode* left = leftIsLeaf ? (BVHNode*)(&leaves[j - 1]) : (BVHNode*)(&inner[j - 1]);
        BVHNode* right = rightIsLeaf ? (BVHNode*)(&leaves[j]) : (BVHNode*)(&inner[j]);
        left->m_parent = inner + i;
        right->m_parent = inner + i;
        inner[i].m_children[0] = left;
        inner[i].m_children[1] = right;
    });

    hipError_t err = hipGetLastError();
    FW_ASSERT(err == hipSuccess);

    return inner; // inner[0] is the root node.
}

FW::BVHNode* FW::BatchSplitBVHBuilder::batchRun(BatchSplitBVHBuilder& BS, AABB& rootBounds)
{
    S32 N = BS.m_bvh.getScene()->getNumTriangles();

    S32 maxN = (S32)(BS.m_params.maxDuplication * (float)N);

    N = 32; // XXX

    FW_ASSERT(BS.m_platform.getTriangleBatchSize() == n_triBatchSize);
    FW_ASSERT(BS.m_platform.getNodeBatchSize() == n_nodeBatchSize);
    FW_ASSERT(BS.m_platform.getSAHTriangleCost() == n_SAHTriangleCost);
    FW_ASSERT(BS.m_platform.getSAHNodeCost() == n_SAHNodeCost);

    FW::Scene* scene = BS.m_bvh.getScene();
    initBBArrays(maxN, scene, BS.m_bvh);

    // Do this in every function that uses these so they can be used in device lambdas
    S32*  refTriIdx = m_refTriIdx;
    AABB* refBounds = m_refBounds;

    const Vec3i* tris = m_tris;
    const Vec3f* verts = m_verts;

    // Determine triangle and root bounds
    rootBounds = thrust::transform_reduce(thrust::device, thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(N), [refTriIdx, refBounds, tris, verts] BHD(S32 i) {
        refTriIdx[i] = i;
        refBounds[i] = AABB();
        for (int j = 0; j < 3; j++)
            refBounds[i].grow(verts[tris[i][j]]);
        return refBounds[i];
    }, AABB(), [] BHD(AABB a, AABB b) { return a + b; });

    // Initialize rest of the members
    m_minOverlap = rootBounds.area() * BS.m_params.splitAlpha;
    //printf("rootBounds: ");
    //rootBounds.print();

    // Build by generation
    S32 nSegments = -1; // Number of nodes (segments), which is the number of unique keys
    for (S32 level = 0; level < 64; level++) {
        doGeneration(N, nSegments, level); // Modifies N
        if (nSegments == N)
            break;
    }

    BVHNode* root = makeNodes(N);

    // OPT: BS.m_bvh.getTriIndices().compact(); Can't do this yet because we can't realloc managed.

    freeArrays();

    return root;
}

FW::BatchSplitBVHBuilder::BatchSplitBVHBuilder(FW::BVH& bvh, const FW::BVH::BuildParams& params)
    : m_bvh(bvh), m_platform(bvh.getPlatform()), m_params(params)
{
}

FW::BatchSplitBVHBuilder::~BatchSplitBVHBuilder(void)
{
}

FW::BVHNode* FW::BatchSplitBVHBuilder::run(void)
{
    printf("BatchSBVH alpha=%g minLeafSize=%d maxLeafSize=%d\n",
        m_params.splitAlpha, m_platform.getMinLeafSize(), m_platform.getMaxLeafSize());

    Timer progressTimer;
    progressTimer.start();

    AABB rootBounds;
    BVHNode* root = batchRun(*this, rootBounds);

    printf("BatchSplitBVHBuilder: t=%f duplicates %.0f%%\n", progressTimer.end(),
        100.0f, (F32)m_numDuplicates / (F32)m_bvh.getScene()->getNumTriangles() * 100.0f);

    // Fix everything up on CPU for now.
    hipDeviceSynchronize();
    root->computeSubtreeValues(m_platform, rootBounds.area(), true, true);

    return root;
}
