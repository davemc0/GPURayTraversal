
#include <hip/hip_runtime.h>
#if 0

#include "base/ManagedAllocator.hpp"
#include <thrust/fill.h>
#include <thrust/logical.h>
#include <thrust/execution_policy.h>
#include <vector>
#include <algorithm>
#include <numeric>
#include <cassert>
#include <iostream>

// create a nickname for vectors which use a managed_allocator
template<class T>
using managed_vector = std::vector<T, managed_allocator<T>>;

__global__ void increment_kernel(int *data, size_t n)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n)
    {
        data[i] += 1;
    }
}

int main()
{
    size_t n = 1 << 20;

    managed_vector<int> vec(n);

    // we can use the vector from the host
    std::iota(vec.begin(), vec.end(), 0);

    std::vector<int> ref(n);
    std::iota(ref.begin(), ref.end(), 0);
    assert(std::equal(ref.begin(), ref.end(), vec.begin()));

    // we can also use it in a CUDA kernel
    size_t block_size = 256;
    size_t num_blocks = (n + (block_size - 1)) / block_size;

    increment_kernel << <num_blocks, block_size >> > (vec.data(), vec.size());

    cudaDeviceSynchronize();

    std::for_each(ref.begin(), ref.end(), [](int& x)
    {
        x += 1;
    });

    assert(std::equal(ref.begin(), ref.end(), vec.begin()));

    // we can also use it with Thrust algorithms

    // by default, the Thrust algorithm will execute on the host with the managed_vector
    thrust::fill(vec.begin(), vec.end(), 7);
    assert(std::all_of(vec.begin(), vec.end(), [](int x)
    {
        return x == 7;
    }));

    // to execute on the device, use the thrust::device execution policy
    thrust::fill(thrust::device, vec.begin(), vec.end(), 13);

    // we need to synchronize before attempting to use the vector on the host
    cudaDeviceSynchronize();

    // to execute on the host, use the thrust::host execution policy
    assert(thrust::all_of(thrust::host, vec.begin(), vec.end(), [](int x)
    {
        return x == 13;
    }));

    std::cout << "OK" << std::endl;

    return 0;
}

#endif

float testThrust(size_t N)
{
    return float(N);
}
